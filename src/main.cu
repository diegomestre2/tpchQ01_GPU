#include "hip/hip_runtime.h"
#include "helper.hpp"
#include "data_types.h"
#include "constants.hpp"
#include "bit_operations.hpp"
#include "kernels/ht_in_global_mem.hpp"
#include "kernels/ht_in_registers.cuh"
#include "kernels/ht_per_thread_in_registers.cuh"
#include "kernels/ht_in_local_mem.cuh"
#include "kernels/ht_per_thread_in_shared_mem.cuh"
// #include "kernels/ht_per_block_in_shared_mem.cuh"
#include "expl_comp_strat/tpch_kit.hpp"
#include "expl_comp_strat/common.hpp"
#include "cpu/common.hpp"
#include "cpu.h"
#include "file_access.hpp"

#include <iostream>
#include <cuda/api_wrappers.h>
#include <vector>
#include <iomanip>
#include <chrono>
#include <unordered_map>
#include <numeric>

#ifndef GPU
#error The GPU preprocessor directive must be defined (ask Tim for the reason)
#endif

using std::tie;
using std::make_pair;
using std::make_unique;
using std::unique_ptr;
using std::cout;
using std::cerr;
using std::endl;
using std::flush;
using std::string;

inline void assert_always(bool a) {
    assert(a);
    if (!a) {
        fprintf(stderr, "Assert always failed!");
        exit(EXIT_FAILURE);
    }
}

using timer = std::chrono::high_resolution_clock;

template <bool Compressed>
struct stream_input_buffer_set;

enum : bool { is_compressed = true, is_not_compressed = false};

template <> struct stream_input_buffer_set<is_compressed> {
    template <typename T> using unique_ptr = cuda::memory::device::unique_ptr<T>;
    unique_ptr< compressed::ship_date_t[]      > ship_date;
    unique_ptr< compressed::discount_t[]       > discount;
    unique_ptr< compressed::extended_price_t[] > extended_price;
    unique_ptr< compressed::tax_t[]            > tax;
    unique_ptr< compressed::quantity_t[]       > quantity;
    unique_ptr< bit_container_t[]              > return_flag;
    unique_ptr< bit_container_t[]              > line_status;
};

template <> struct stream_input_buffer_set<is_not_compressed> {
    template <typename T> using unique_ptr = cuda::memory::device::unique_ptr<T>;
    unique_ptr< ship_date_t[]      > ship_date;
    unique_ptr< discount_t[]       > discount;
    unique_ptr< extended_price_t[] > extended_price;
    unique_ptr< tax_t[]            > tax;
    unique_ptr< quantity_t[]       > quantity;
    unique_ptr< return_flag_t[]    > return_flag;
    unique_ptr< line_status_t[]    > line_status;
};


// Note: This will force casts to int. It's not a problem
// the way our code is written, but otherwise it needs to be generalized
constexpr inline int div_rounding_up(const int& dividend, const int& divisor)
{
    // This is not the fastest implementation, but it's safe, in that there's never overflow
#if __cplusplus >= 201402L
    std::div_t div_result = std::div(dividend, divisor);
    return div_result.quot + !(!div_result.rem);
#else
    // Hopefully the compiler will optimize the two calls away.
    return std::div(dividend, divisor).quot + !(!std::div(dividend, divisor).rem);
#endif
}

void print_help(int argc, char** argv) {
    fprintf(stderr, "Unrecognized command line option.\n");
    fprintf(stderr, "Usage: %s [args]\n", argv[0]);
    fprintf(stderr, "   --apply-compression\n");
    fprintf(stderr, "   --print-results\n");
    fprintf(stderr, "   --use-filter-pushdown\n");
    fprintf(stderr, "   --use-coprocessing (currently ignored)\n");
    fprintf(stderr, "   --hash-table-placement=[default:in-registers-per-thread]\n"
                    "     (one of: in-registers, in-registers-per-thread, local-mem, per-thread-shared-mem, global))\n");
    fprintf(stderr, "   --sf=[default:%f] (number, e.g. 0.01 - 100)\n", defaults::scale_factor);
    fprintf(stderr, "   --streams=[default:%u] (number, e.g. 1 - 64)\n", defaults::num_gpu_streams);
    fprintf(stderr, "   --threads-per-block=[default:%u] (number, e.g. 32 - 1024)\n", defaults::num_threads_per_block);
    fprintf(stderr, "   --tuples-per-thread=[default:%u] (number, e.g. 1 - 1048576)\n", defaults::num_tuples_per_thread);
    fprintf(stderr, "   --tuples-per-kernel=[default:%u] (number, e.g. 64 - 4194304)\n", defaults::num_tuples_per_kernel_launch);
}

template <typename F, typename... Args>
void for_each_argument(F f, Args&&... args) {
    [](...){}((f(std::forward<Args>(args)), 0)...);
}

void make_sure_we_are_on_cpu_core_0()
{
#if 0
    // CPU affinities are devil's work
    // Make sure we are on core 0
    // TODO: Why not in a function?
    cpu_set_t cpuset; 

    CPU_ZERO(&cpuset);
    CPU_SET(0, &cpuset);
    sched_setaffinity(0, sizeof(cpuset), &cpuset);
#endif
}

std::pair<string,string> split_once(string delimited, char delimiter) {
    auto pos = delimited.find_first_of(delimiter);
    return { delimited.substr(0, pos), delimited.substr(pos+1) };
}

template <typename T>
void print_results(const T& aggregates_on_host, cardinality_t cardinality) {
    cout << "+---------------------------------------------------- Results ------------------------------------------------------+\n";
    cout << "|  LS | RF |  sum_quantity        |  sum_base_price      |  sum_disc_price      |  sum_charge          | count      |\n";
    cout << "+-------------------------------------------------------------------------------------------------------------------+\n";
    auto print_dec = [] (auto s, auto x) { printf("%s%17ld.%02ld", s, Decimal64::GetInt(x), Decimal64::GetFrac(x)); };
    cardinality_t total_passing { 0 };

    for (int group=0; group<num_potential_groups; group++) {
        if (true) { // (aggregates_on_host.record_count[group] > 0) {
            char rf = decode_return_flag(group >> line_status_bits);
            char ls = decode_line_status(group & 0b1);
            if (rf == 'A' and ls == 'F') {
                if (cardinality == 6001215) {
                    assert(aggregates_on_host.sum_quantity[group] == 3773410700);
                    assert(aggregates_on_host.record_count[group] == 1478493);
                }
            } else if (rf == 'N' and ls == 'F') {
                if (cardinality == 6001215) {
                    assert(aggregates_on_host.sum_quantity[group] == 99141700);
                    assert(aggregates_on_host.record_count[group] == 38854);
                }
            } else if (rf == 'N' and ls == 'O') {
                rf = 'N';
                ls = 'O';
                if (cardinality == 6001215) {
                    assert(aggregates_on_host.sum_quantity[group] == 7447604000);
                    assert(aggregates_on_host.record_count[group] == 2920374);
                }
            } else if (rf == 'R' and ls == 'F') {
                if (cardinality == 6001215) {
                    assert(aggregates_on_host.sum_quantity[group]== 3771975300);
                    assert(aggregates_on_host.record_count[group]== 1478870);
                }
            }

            printf("| # %c | %c ", rf, ls);
            print_dec(" | ",  aggregates_on_host.sum_quantity.get()[group]);
            print_dec(" | ",  aggregates_on_host.sum_base_price.get()[group]);
            print_dec(" | ",  aggregates_on_host.sum_discounted_price.get()[group]);
            print_dec(" | ",  aggregates_on_host.sum_charge.get()[group]);
            printf(" | %10u |\n", aggregates_on_host.record_count.get()[group]);
            total_passing += aggregates_on_host.record_count.get()[group];
        }
    }
    cout << "+-------------------------------------------------------------------------------------------------------------------+\n";
    cout << "Total number of elements tuples satisfying the WHERE clause: " << total_passing << "\n";
}

const std::unordered_map<string, cuda::device_function_t> kernels = {
    { "local-mem",               cuda::in_local_mem_ht_tpchQ01            },
    { "in-registers",            cuda::in_registers_ht_tpchQ01            },
    { "in-registers-per-thread", cuda::in_registers_per_thread_ht_tpchQ01 },
    { "per-thread-shared-mem",   cuda::thread_in_shared_mem_ht_tpchQ01<>  },
//  { "per-block-shared-mem",    cuda::shared_mem_ht_tpchQ01              },
    { "global",                  cuda::global_ht_tpchQ01                  },
};

const std::unordered_map<string, cuda::device_function_t> kernels_compressed = {
    { "local-mem",               cuda::in_local_mem_ht_tpchQ01_compressed            },
    { "in-registers",            cuda::in_registers_ht_tpchQ01_compressed            },
    { "in-registers-per-thread", cuda::in_registers_per_thread_ht_tpchQ01_compressed },
    { "per-thread-shared-mem",   cuda::thread_in_shared_mem_ht_tpchQ01_compressed<>  },
//  { "per-block-shared-mem",    cuda::shared_mem_ht_tpchQ01_compressed              },
    { "global",                  cuda::global_ht_tpchQ01_compressed                  },
};

const std::unordered_map<string, cuda::device_function_t> kernels_filter_pushdown = {
    { "local-mem",               cuda::in_local_mem_ht_tpchQ01_filter_pushdown_compressed            },
    { "in-registers",            cuda::in_registers_ht_tpchQ01_filter_pushdown_compressed            },
    { "in-registers-per-thread", cuda::in_registers_per_thread_ht_tpchQ01_filter_pushdown_compressed },
    { "global",                  cuda::global_ht_tpchQ01_filter_pushdown_compressed                  },
//  { "per-block-shared-mem",    cuda::shared_mem_ht_tpchQ01_pushdown_compressed                     },
    { "per-thread-shared-mem",   cuda::thread_in_shared_mem_ht_tpchQ01_pushdown_compressed<>         },
};

// Some kernel variants cannot support as many threads per block as the hardware allows generally,
// and for these we use a fixed number for now
const std::unordered_map<string, cuda::grid_block_dimension_t> fixed_threads_per_block = {
    { "per-thread-shared-mem", cuda::max_threads_per_block_for_per_thread_shared_mem },
    { "in-registers",          cuda::threads_per_block_for_in_registers_hash_table },
};

const std::unordered_map<string, unsigned> num_threads_to_handle_tuple = {
    { "local-mem",               1  },
    { "in-registers",            div_rounding_up(warp_size, warp_size / num_potential_groups)  },
    { "in-registers-per-thread", 1  },
    { "per-thread-shared-mem",   1  },
//  { "per-block-shared-mem",    1? },
    { "global",                  1  },
};

struct q1_params_t {

	// Command-line-settable parameters

	double scale_factor                  { defaults::scale_factor };
    std::string kernel_variant           { defaults::kernel_variant };
    bool should_print_results            { defaults::should_print_results };
    bool use_filter_pushdown             { false };
    bool apply_compression               { defaults::apply_compression };
    int num_gpu_streams                  { defaults::num_gpu_streams };
    cuda::grid_block_dimension_t num_threads_per_block
                                         { defaults::num_threads_per_block };
    int num_tuples_per_thread            { defaults::num_tuples_per_thread };

    int num_tuples_per_kernel_launch     { defaults::num_tuples_per_kernel_launch };
        // Make sure it's a multiple of num_threads_per_block and of warp_size, or bad things may happen

	// This is the number of times we run the actual query execution - the part that we time;
    // it will not include initialization/allocations that are not necessary when the DBMS
    // is brought up. Note the allocation vs sub-allocation issue (see further comments below)
    int num_query_execution_runs         { defaults::num_query_execution_runs };

    bool use_coprocessing                { false };
    bool user_set_num_threads_per_block  { false };
};

q1_params_t parse_command_line(int argc, char** argv)
{
	q1_params_t params;

    for(int i = 1; i < argc; i++) {
        auto arg = string(argv[i]);
        if (arg.substr(0,2) != "--") {
            print_help(argc, argv);
            exit(EXIT_FAILURE);
        }
        arg = arg.substr(2);
        if (arg == "device") {
            get_device_properties();
            exit(1);
        } else if (arg == "use-coprocessing") {
            params.use_coprocessing = true;
        } else if (arg == "apply-compression") {
        	params.apply_compression = true;
        } else if (arg == "use-filter-pushdown") {
        	params.use_filter_pushdown = true;
        	params.apply_compression = true;
        }  else if (arg == "print-results") {
        	params.should_print_results = true;
        } else {
            // A  name=value argument
            auto p = split_once(arg, '=');
            auto& arg_name = p.first; auto& arg_value = p.second;
            if (arg_name == "scale-factor") {
            	params.scale_factor = std::stod(arg_value);
                if (params.scale_factor - 0 < 0.001) {
                    cerr << "Invalid scale factor " + std::to_string(params.scale_factor) << endl;
                    exit(EXIT_FAILURE);
                }
            } else if (arg_name == "hash-table-placement") {
            	params.kernel_variant = arg_value;
                if (kernels.find(params.kernel_variant) == kernels.end()) {
                    cerr << "No kernel variant named \"" + params.kernel_variant + "\" is available" << endl;
                    exit(EXIT_FAILURE);
                }
            } else if (arg_name == "streams") {
            	params.num_gpu_streams = std::stoi(arg_value);
            } else if (arg_name == "tuples-per-thread") {
            	params.num_tuples_per_thread = std::stoi(arg_value);
            } else if (arg_name == "threads-per-block") {
            	params.num_threads_per_block = std::stoi(arg_value);
                params.user_set_num_threads_per_block = true;
            } else if (arg_name == "tuples-per-kernel-launch") {
            	params.num_tuples_per_kernel_launch = std::stoi(arg_value);
            } else if (arg_name == "runs") {
            	params.num_query_execution_runs = std::stoi(arg_value);
                if (params.num_query_execution_runs <= 0) {
                    cerr << "Number of runs must be positive" << endl;
                    exit(EXIT_FAILURE);
                }
            } else {
                print_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        }
    }
    if (fixed_threads_per_block.find(params.kernel_variant) != fixed_threads_per_block.end()) {
        if (params.user_set_num_threads_per_block and
            (fixed_threads_per_block.at(params.kernel_variant) != params.num_threads_per_block)) {
            throw std::invalid_argument("Invalid number of threads per block for kernel variant "
                + params.kernel_variant + " (it must be "
                + std::to_string(fixed_threads_per_block.at(params.kernel_variant)) + ")");
        }
        params.num_threads_per_block = fixed_threads_per_block.at(params.kernel_variant);
    }
    return params;
}

int main(int argc, char** argv) {
    cout << "TPC-H Query 1" << '\n';
    make_sure_we_are_on_cpu_core_0();

    auto params = parse_command_line(argc, argv);
    cardinality_t cardinality; // This is computed rather than being set manually


    lineitem li((size_t)(7000000 * std::max(params.scale_factor, 1.0)));
        // TODO: lineitem should really not need this cap, it should just adjust
        // allocated space as the need arises (and start with an estimate based on
        // the file size

    std::unique_ptr< ship_date_t[]      > _shipdate;
    std::unique_ptr< return_flag_t[]    > _returnflag;
    std::unique_ptr< line_status_t[]    > _linestatus;
    std::unique_ptr< discount_t[]       > _discount;
    std::unique_ptr< tax_t[]            > _tax;
    std::unique_ptr< extended_price_t[] > _extendedprice;
    std::unique_ptr< quantity_t[]       > _quantity;

    auto data_files_directory =
        filesystem::path(defaults::tpch_data_subdirectory) / std::to_string(params.scale_factor);
    auto parsed_columns_are_cached =
        filesystem::exists(data_files_directory / "shipdate.bin");
    if (parsed_columns_are_cached) {
        // binary files (seem to) exist, load them
        cardinality = filesystem::file_size(data_files_directory / "shipdate.bin") / sizeof(ship_date_t);
        if (cardinality == cardinality_of_scale_factor_1) {
            cardinality = ((double) cardinality) * params.scale_factor;
        }
        cout << "Lineitem table cardinality for scale factor " << params.scale_factor << " is " << cardinality << endl;
        if (cardinality == 0) {
            throw std::runtime_error("The lineitem table column cardinality should not be 0");
        }
        load_column_from_binary_file(_shipdate,      cardinality, data_files_directory, "shipdate.bin");
        load_column_from_binary_file(_returnflag,    cardinality, data_files_directory, "returnflag.bin");
        load_column_from_binary_file(_linestatus,    cardinality, data_files_directory, "linestatus.bin");
        load_column_from_binary_file(_discount,      cardinality, data_files_directory, "discount.bin");
        load_column_from_binary_file(_tax,           cardinality, data_files_directory, "tax.bin");
        load_column_from_binary_file(_extendedprice, cardinality, data_files_directory, "extendedprice.bin");
        load_column_from_binary_file(_quantity,      cardinality, data_files_directory, "quantity.bin");

        // See: We don't need no stinkin' macros these days. Actually, we can do something
        // similar with a lot of the replicated code in this file
        for_each_argument(
            [&](auto tup){
                std::get<0>(tup).cardinality = cardinality;
                std::get<0>(tup).m_ptr = std::get<1>(tup).get();
            },
            tie(li.l_shipdate,      _shipdate),
            tie(li.l_returnflag,    _returnflag),
            tie(li.l_linestatus,    _linestatus),
            tie(li.l_discount,      _discount),
            tie(li.l_tax,           _tax),
            tie(li.l_extendedprice, _extendedprice), 
            tie(li.l_quantity,      _quantity)
        );
    } else {
        // TODO: Take this out into a script

        filesystem::create_directory(defaults::tpch_data_subdirectory);
        filesystem::create_directory(data_files_directory);
        auto table_file_path = data_files_directory / lineitem_table_file_name;
        if (not filesystem::exists(table_file_path)) {
            throw std::runtime_error("Cannot locate table text file " + table_file_path.string());
            // Not generating it ourselves - that's: 1. Not healthy and 2. Not portable;
            // setup scripts are intended to do that
        }
        cout << "Parsing the lineitem table in file " << table_file_path << endl;
        li.FromFile(table_file_path.c_str());
        cardinality = li.l_extendedprice.cardinality;
        if (cardinality == cardinality_of_scale_factor_1) {
            cardinality = ((double) cardinality) * params.scale_factor;
        }
        if (cardinality == 0) {
            throw std::runtime_error("The lineitem table column cardinality should not be 0");
        }
        cout << "CSV read & parsed; table length: " << cardinality << " records." << endl;
        auto write_to = [&](auto& uptr, const char* filename) {
            using T = typename std::remove_pointer<typename std::decay<decltype(uptr.get())>::type>::type;
            load_column_from_binary_file(uptr.get(), cardinality, data_files_directory, "shipdate.bin");
        };
        write_column_to_binary_file(li.l_shipdate.get(),      cardinality, data_files_directory, "shipdate.bin");
        write_column_to_binary_file(li.l_returnflag.get(),    cardinality, data_files_directory, "returnflag.bin");
        write_column_to_binary_file(li.l_linestatus.get(),    cardinality, data_files_directory, "linestatus.bin");
        write_column_to_binary_file(li.l_discount.get(),      cardinality, data_files_directory, "discount.bin");
        write_column_to_binary_file(li.l_tax.get(),           cardinality, data_files_directory, "tax.bin");
        write_column_to_binary_file(li.l_extendedprice.get(), cardinality, data_files_directory, "extendedprice.bin");
        write_column_to_binary_file(li.l_quantity.get(),      cardinality, data_files_directory, "quantity.bin");
    }

    CoProc* cpu_coprocessor = params.use_coprocessing ?  new CoProc(li, true) : nullptr;

    auto compressed_ship_date      = cuda::memory::host::make_unique< compressed::ship_date_t[]      >(cardinality);
    auto compressed_discount       = cuda::memory::host::make_unique< compressed::discount_t[]       >(cardinality);
    auto compressed_extended_price = cuda::memory::host::make_unique< compressed::extended_price_t[] >(cardinality);
    auto compressed_tax            = cuda::memory::host::make_unique< compressed::tax_t[]            >(cardinality);
    auto compressed_quantity       = cuda::memory::host::make_unique< compressed::quantity_t[]       >(cardinality);

    auto compressed_return_flag    = cuda::memory::host::make_unique< bit_container_t[] >(div_rounding_up(cardinality, return_flag_values_per_container));
    auto compressed_line_status    = cuda::memory::host::make_unique< bit_container_t[] >(div_rounding_up(cardinality, line_status_values_per_container));

    auto ship_date_bit_vector      = cuda::memory::host::make_unique< uint8_t[]         >(div_rounding_up(cardinality, 8));

    auto ship_date      = li.l_shipdate.get();
    auto return_flag    = li.l_returnflag.get();
    auto line_status    = li.l_linestatus.get();
    auto discount       = li.l_discount.get();
    auto tax            = li.l_tax.get();
    auto extended_price = li.l_extendedprice.get();
    auto quantity       = li.l_quantity.get();

    if (params.apply_compression) {
        cout << "Preprocessing/compressing column data... " << flush;

        // Man, we really need to have a sub-byte-length-value container class
        std::memset(compressed_return_flag.get(), 0, div_rounding_up(cardinality, return_flag_values_per_container));
        std::memset(compressed_line_status.get(), 0, div_rounding_up(cardinality, line_status_values_per_container));
        for(cardinality_t i = 0; i < cardinality; i++) {
            compressed_ship_date[i]      = ship_date[i] - ship_date_frame_of_reference;
            compressed_discount[i]       = discount[i]; // we're keeping the factor 100 scaling
            compressed_extended_price[i] = extended_price[i];
            compressed_quantity[i]       = quantity[i] / 100;
            compressed_tax[i]            = tax[i]; // we're keeping the factor 100 scaling
            set_bit_resolution_element<log_return_flag_bits, cardinality_t>(
                compressed_return_flag.get(), i, encode_return_flag(return_flag[i]));
            set_bit_resolution_element<log_line_status_bits, cardinality_t>(
                compressed_line_status.get(), i, encode_line_status(line_status[i]));
            assert( (ship_date_t)      compressed_ship_date[i]      == ship_date[i] - ship_date_frame_of_reference);
            assert( (discount_t)       compressed_discount[i]       == discount[i]);
            assert( (extended_price_t) compressed_extended_price[i] == extended_price[i]);
            assert( (quantity_t)       compressed_quantity[i]       == quantity[i] / 100);
                // not keeping the scaling here since we know the data is all integral; you could call this a form
                // of compression
            assert( (tax_t)            compressed_tax[i]            == tax[i]);
        }
        for(cardinality_t i = 0; i < cardinality; i++) {
            assert(decode_return_flag(get_bit_resolution_element<log_return_flag_bits, cardinality_t>(compressed_return_flag.get(), i)) == return_flag[i]);
            assert(decode_line_status(get_bit_resolution_element<log_line_status_bits, cardinality_t>(compressed_line_status.get(), i)) == line_status[i]);
        }

        cout << "done." << endl;
    }

    // Note:
    // We are not timing the host-side allocations here. In a real DBMS, these will likely only be
    // a few sub-allocations, which would take very little time (dozens of clock cycles overall) -
    // no system calls.

    struct {
        std::unique_ptr<sum_quantity_t[]        > sum_quantity;
        std::unique_ptr<sum_base_price_t[]      > sum_base_price;
        std::unique_ptr<sum_discounted_price_t[]> sum_discounted_price;
        std::unique_ptr<sum_charge_t[]          > sum_charge;
        std::unique_ptr<sum_discount_t[]        > sum_discount;
        std::unique_ptr<cardinality_t[]         > record_count;
        // Why aren't we computing these? They're part of TPC-H Q1 after all
        // struct {
        //     std::unique_ptr<avg_quantity_t[]        > avg_quantity;
        //     std::unique_ptr<avg_extended_price_t[]  > avg_extended_price;
        //     std::unique_ptr<avg_discount_t[]        > avg_discount;
        // } derived;
    } aggregates_on_host = {
        std::make_unique< sum_quantity_t[]         >(num_potential_groups),
        std::make_unique< sum_base_price_t[]       >(num_potential_groups),
        std::make_unique< sum_discounted_price_t[] >(num_potential_groups),
        std::make_unique< sum_charge_t []          >(num_potential_groups),
        std::make_unique< sum_discount_t[]         >(num_potential_groups),
        std::make_unique< cardinality_t[]          >(num_potential_groups)
        // ,
        // {
        //      std::make_unique< avg_quantity_t[]         >(num_potential_groups),
        //      std::make_unique< avg_extended_price_t[]   >(num_potential_groups),
        //      std::make_unique< avg_discount_t[]         >(num_potential_groups),
        // }
    };

    /* Allocate memory on device */
    
    // Note:
    // We are not timing the allocations here. In a real DBMS, actual CUDA allocations would
    // happen with the DBMS is brought up, and when a query is processed, it will only be
    // a few sub-allocations, which would take very little time (dozens of clock cycles overall) -
    // no CUDA API nor system calls. We _will_, however, time the initialization of the buffers.

    auto cuda_device = cuda::device::current::get();

    struct {
        cuda::memory::device::unique_ptr< sum_quantity_t[]         > sum_quantity;
        cuda::memory::device::unique_ptr< sum_base_price_t[]       > sum_base_price;
        cuda::memory::device::unique_ptr< sum_discounted_price_t[] > sum_discounted_price;
        cuda::memory::device::unique_ptr< sum_charge_t[]           > sum_charge;
        cuda::memory::device::unique_ptr< sum_discount_t[]         > sum_discount;
        cuda::memory::device::unique_ptr< cardinality_t[]          > record_count;
    } aggregates_on_device = {
        cuda::memory::device::make_unique< sum_quantity_t[]         >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_base_price_t[]       >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_discounted_price_t[] >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_charge_t []          >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_discount_t[]         >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< cardinality_t[]          >(cuda_device, num_potential_groups)
    };

    struct {
        std::vector<stream_input_buffer_set<is_not_compressed> > uncompressed;
        std::vector<stream_input_buffer_set<is_compressed    > > compressed;
    } stream_input_buffer_sets;
    std::vector<cuda::stream_t<>> streams;
    if (params.apply_compression) {
        stream_input_buffer_sets.compressed.reserve(params.num_gpu_streams);
    } else {
        stream_input_buffer_sets.uncompressed.reserve(params.num_gpu_streams);
    }
    streams.reserve(params.num_gpu_streams);
        // We'll be scheduling (most of) our work in a round-robin fashion on all of
        // the streams, to prevent the GPU from idling.


    for (int i = 0; i < params.num_gpu_streams; ++i) {
        if (params.apply_compression) {
            auto input_buffers = stream_input_buffer_set<is_compressed>{
                cuda::memory::device::make_unique< compressed::ship_date_t[]      >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< compressed::discount_t[]       >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< compressed::extended_price_t[] >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< compressed::tax_t[]            >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< compressed::quantity_t[]       >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< bit_container_t[]              >(cuda_device, div_rounding_up(params.num_tuples_per_kernel_launch, return_flag_values_per_container)),
                cuda::memory::device::make_unique< bit_container_t[]              >(cuda_device, div_rounding_up(params.num_tuples_per_kernel_launch, line_status_values_per_container))
            };
            stream_input_buffer_sets.compressed.emplace_back(std::move(input_buffers));
        }
        else {
            auto input_buffers = stream_input_buffer_set<is_not_compressed>{
                cuda::memory::device::make_unique< ship_date_t[]      >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< discount_t[]       >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< extended_price_t[] >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< tax_t[]            >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< quantity_t[]       >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< return_flag_t[]    >(cuda_device, params.num_tuples_per_kernel_launch),
                cuda::memory::device::make_unique< line_status_t[]    >(cuda_device, params.num_tuples_per_kernel_launch),
            };
            stream_input_buffer_sets.uncompressed.emplace_back(std::move(input_buffers));
        }
        auto stream = cuda_device.create_stream(cuda::stream::async);
        streams.emplace_back(std::move(stream));
    }

    // You can't measure this from inside the process - without events, which
    // double copy_time = 0;
    // double computation_time = 0;

    // This only works for the overall time, not for anything else, so it's not a good idea:
     std::ofstream results_file;
     results_file.open("results.csv", std::ios::out);

     cuda::profiling::start();

    for(int run_index = 0; run_index < params.num_query_execution_runs; run_index++) {
        cout << "Executing query, run " << run_index + 1 << " of " << params.num_query_execution_runs << "... " << flush;
        if (params.use_coprocessing) {
             cpu_coprocessor->Clear();
        }
        auto start = timer::now();
        
        auto gpu_end_offset = cardinality;
        if (params.use_coprocessing) {
             // Split the work between the CPU and the GPU at 50% each
             // TODO: 
             // - Double-check the choice of alignment here
             // - The parameters here are weird
             auto cpu_start_offset = cardinality - cardinality / 20;
             cpu_start_offset = cpu_start_offset - cpu_start_offset % params.num_tuples_per_kernel_launch;
             auto num_records_for_cpu_to_process = cardinality - cpu_start_offset;
             (*cpu_coprocessor)(cpu_start_offset, num_records_for_cpu_to_process);
             gpu_end_offset = cpu_start_offset;
        } 

        // Initialize the aggregates; perhaps we should do this in a single kernel? ... probably not worth it
        streams[0].enqueue.memset(aggregates_on_device.sum_quantity.get(),         0, num_potential_groups * sizeof(sum_quantity_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_base_price.get(),       0, num_potential_groups * sizeof(sum_base_price_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_discounted_price.get(), 0, num_potential_groups * sizeof(sum_discounted_price_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_charge.get(),           0, num_potential_groups * sizeof(sum_charge_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_discount.get(),         0, num_potential_groups * sizeof(sum_discount_t));
        streams[0].enqueue.memset(aggregates_on_device.record_count.get(),         0, num_potential_groups * sizeof(cardinality_t));

        cuda::event_t aggregates_initialized_event = streams[0].enqueue.event(
            cuda::event::sync_by_blocking, cuda::event::dont_record_timings, cuda::event::not_interprocess);
        for (int i = 1; i < params.num_gpu_streams; ++i) {
            streams[i].enqueue.wait(aggregates_initialized_event);
            // The other streams also require the aggregates to be initialized before doing any work
        }
        auto stream_index = 0;
        for (size_t offset_in_table = 0;
             offset_in_table < gpu_end_offset;
             offset_in_table += params.num_tuples_per_kernel_launch,
             stream_index = (stream_index+1) % params.num_gpu_streams)
        {
            auto num_tuples_for_this_launch = std::min<cardinality_t>(params.num_tuples_per_kernel_launch, gpu_end_offset - offset_in_table);
            auto num_return_flag_bit_containers_for_this_launch = div_rounding_up(num_tuples_for_this_launch, return_flag_values_per_container);
            auto num_line_status_bit_containers_for_this_launch = div_rounding_up(num_tuples_for_this_launch, line_status_values_per_container);

            // auto start_copy = timer::now();  // This can't work, since copying is asynchronous.
            auto& stream = streams[stream_index];

            if (params.apply_compression) {
                auto& input_buffers = stream_input_buffer_sets.compressed[stream_index];
                stream.enqueue.copy(input_buffers.discount.get()      , compressed_discount.get()       + offset_in_table, num_tuples_for_this_launch * sizeof(compressed::discount_t));
                stream.enqueue.copy(input_buffers.extended_price.get(), compressed_extended_price.get() + offset_in_table, num_tuples_for_this_launch * sizeof(compressed::extended_price_t));
                stream.enqueue.copy(input_buffers.tax.get()           , compressed_tax.get()            + offset_in_table, num_tuples_for_this_launch * sizeof(compressed::tax_t));
                stream.enqueue.copy(input_buffers.quantity.get()      , compressed_quantity.get()       + offset_in_table, num_tuples_for_this_launch * sizeof(compressed::quantity_t));
                stream.enqueue.copy(input_buffers.return_flag.get()   , compressed_return_flag.get()    + offset_in_table / return_flag_values_per_container, num_return_flag_bit_containers_for_this_launch * sizeof(bit_container_t));
                stream.enqueue.copy(input_buffers.line_status.get()   , compressed_line_status.get()    + offset_in_table / line_status_values_per_container, num_line_status_bit_containers_for_this_launch * sizeof(bit_container_t));
                if (params.use_filter_pushdown) {
                    cuda::profiling::scoped_range_marker("on-CPU filtering");
                    auto shipdate_bit_vector = ship_date_bit_vector.get();
                    auto shipdate_compressed = compressed_ship_date.get();
                    size_t target = offset_in_table + num_tuples_for_this_launch;
                    for(size_t i = offset_in_table; i < target; i += 8) {
                        shipdate_bit_vector[i / 8] = 0;
                        for(size_t j = 0; j < std::min((size_t) 8, target - i); j++) {
                            shipdate_bit_vector[i / 8] |= (shipdate_compressed[i + j] < compressed_threshold_ship_date) << j;
                        }
                    }
                    stream.enqueue.copy(input_buffers.ship_date.get()     , shipdate_bit_vector             + offset_in_table / 8, ((num_tuples_for_this_launch + 7) / 8) * sizeof(uint8_t));
                } else {
                    stream.enqueue.copy(input_buffers.ship_date.get()     , compressed_ship_date.get()      + offset_in_table, num_tuples_for_this_launch * sizeof(compressed::ship_date_t));
                }
            }
            else {
                auto& input_buffers = stream_input_buffer_sets.uncompressed[stream_index];
                stream.enqueue.copy(input_buffers.ship_date.get()     , ship_date      + offset_in_table, num_tuples_for_this_launch * sizeof(ship_date_t));
                stream.enqueue.copy(input_buffers.discount.get()      , discount       + offset_in_table, num_tuples_for_this_launch * sizeof(discount_t));
                stream.enqueue.copy(input_buffers.extended_price.get(), extended_price + offset_in_table, num_tuples_for_this_launch * sizeof(extended_price_t));
                stream.enqueue.copy(input_buffers.tax.get()           , tax            + offset_in_table, num_tuples_for_this_launch * sizeof(tax_t));
                stream.enqueue.copy(input_buffers.quantity.get()      , quantity       + offset_in_table, num_tuples_for_this_launch * sizeof(quantity_t));
                stream.enqueue.copy(input_buffers.return_flag.get()   , return_flag    + offset_in_table, num_tuples_for_this_launch * sizeof(return_flag_t));
                stream.enqueue.copy(input_buffers.line_status.get()   , line_status    + offset_in_table, num_tuples_for_this_launch * sizeof(line_status_t));
            }

            cuda::grid_block_dimension_t num_threads_per_block;
            cuda::grid_dimension_t       num_blocks;

            if (params.kernel_variant == "in-registers") {
        		auto num_warps_per_block = params.num_threads_per_block / warp_size;
        			// rounding down the number of threads per block!
        		num_threads_per_block = num_warps_per_block * warp_size;
        		auto num_tables_per_warp       = cuda::warp_size / num_potential_groups;
        		auto num_tuples_handled_by_block = num_tables_per_warp * num_warps_per_block * params.num_tuples_per_thread;
                num_blocks = div_rounding_up(
                    num_tuples_for_this_launch,
                    num_tuples_handled_by_block);
            }
            else {
                num_blocks = div_rounding_up(
                        num_tuples_for_this_launch,
                        params.num_threads_per_block * params.num_tuples_per_thread);
                // NOTE: If the number of blocks drops below the number of GPU cores, this is definitely useless,
                // and to be on the safe side - twice as many.
                num_threads_per_block = params.num_threads_per_block;
            }
            auto launch_config = cuda::make_launch_config(num_blocks, num_threads_per_block);
            // cout << "num_tuples_for_this_launch = " << num_tuples_for_this_launch << ", num_blocks = " << num_blocks << ", params.num_threads_per_block = " << params.num_threads_per_block << endl;

            
            if (params.use_filter_pushdown) {
                auto& input_buffers = stream_input_buffer_sets.compressed[stream_index];
                auto kernel = kernels_filter_pushdown.at(params.kernel_variant);
                stream.enqueue.kernel_launch(
                    kernel,
                    launch_config,
                    aggregates_on_device.sum_quantity.get(),
                    aggregates_on_device.sum_base_price.get(),
                    aggregates_on_device.sum_discounted_price.get(),
                    aggregates_on_device.sum_charge.get(),
                    aggregates_on_device.sum_discount.get(),
                    aggregates_on_device.record_count.get(),
                    input_buffers.ship_date.get(),
                    input_buffers.discount.get(),
                    input_buffers.extended_price.get(),
                    input_buffers.tax.get(),
                    input_buffers.quantity.get(),
                    input_buffers.return_flag.get(),
                    input_buffers.line_status.get(),
                    num_tuples_for_this_launch);
            } else if (params.apply_compression) {
                auto& input_buffers = stream_input_buffer_sets.compressed[stream_index];
                auto kernel = kernels_compressed.at(params.kernel_variant);
                stream.enqueue.kernel_launch(
                    kernel,
                    launch_config,
                    aggregates_on_device.sum_quantity.get(),
                    aggregates_on_device.sum_base_price.get(),
                    aggregates_on_device.sum_discounted_price.get(),
                    aggregates_on_device.sum_charge.get(),
                    aggregates_on_device.sum_discount.get(),
                    aggregates_on_device.record_count.get(),
                    input_buffers.ship_date.get(),
                    input_buffers.discount.get(),
                    input_buffers.extended_price.get(),
                    input_buffers.tax.get(),
                    input_buffers.quantity.get(),
                    input_buffers.return_flag.get(),
                    input_buffers.line_status.get(),
                    num_tuples_for_this_launch);
            } else {
                auto& input_buffers = stream_input_buffer_sets.uncompressed[stream_index];
                auto kernel = kernels.at(params.kernel_variant);
                stream.enqueue.kernel_launch(
                    kernel,
                    launch_config,
                    aggregates_on_device.sum_quantity.get(),
                    aggregates_on_device.sum_base_price.get(),
                    aggregates_on_device.sum_discounted_price.get(),
                    aggregates_on_device.sum_charge.get(),
                    aggregates_on_device.sum_discount.get(),
                    aggregates_on_device.record_count.get(),
                    input_buffers.ship_date.get(),
                    input_buffers.discount.get(),
                    input_buffers.extended_price.get(),
                    input_buffers.tax.get(),
                    input_buffers.quantity.get(),
                    input_buffers.return_flag.get(),
                    input_buffers.line_status.get(),
                    num_tuples_for_this_launch);
            }
        }
        std::vector<cuda::event_t> completion_events;
        for(int i = 1; i < params.num_gpu_streams; i++) {
            auto event = streams[i].enqueue.event();
            completion_events.emplace_back(std::move(event));
        }
        
        // It's probably a better idea to go round-robin on the streams here
        streams[0].enqueue.copy(aggregates_on_host.sum_quantity.get(),         aggregates_on_device.sum_quantity.get(),         num_potential_groups * sizeof(sum_quantity_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_base_price.get(),       aggregates_on_device.sum_base_price.get(),       num_potential_groups * sizeof(sum_base_price_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_discounted_price.get(), aggregates_on_device.sum_discounted_price.get(), num_potential_groups * sizeof(sum_discounted_price_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_charge.get(),           aggregates_on_device.sum_charge.get(),           num_potential_groups * sizeof(sum_charge_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_discount.get(),         aggregates_on_device.sum_discount.get(),         num_potential_groups * sizeof(sum_discount_t));
        streams[0].enqueue.copy(aggregates_on_host.record_count.get(),         aggregates_on_device.record_count.get(),         num_potential_groups * sizeof(cardinality_t));

        // TODO: There's some sort of result stability issue here
/*
        for(int i = 1; i < params.num_gpu_streams; i++) {
            streams[i].synchronize();
        }
*/
        streams[0].synchronize();

        if (cpu_coprocessor) { cpu_coprocessor->wait(); }

        auto end = timer::now();

        std::chrono::duration<double> duration(end - start);
        cout << "done." << endl;
        results_file << duration.count() << '\n';
        if (cpu_coprocessor) { 
			assert_always(cpu_coprocessor->numExtantGroups() == 4); 
				// Actually, for scale factors under, say, 0.001, this
				// may realistically end up being 3 instead of 4
		}
        if (params.should_print_results) {
            print_results(aggregates_on_host, cardinality);
        }
    }
    cuda::profiling::stop();
    results_file.close();
}
