#include "hip/hip_runtime.h"
#include <iostream>
#include <cuda/api_wrappers.h>
#include <vector>
#include <iomanip>
#include <fstream>
#include <chrono>

#include "data_types.h"
#include "constants.hpp"
#include "bit_operations.h"
#include "kernel.hpp"
//#include "kernels/naive.hpp"
//#include "kernels/local.hpp"
//#include "kernels/global.hpp"
#include "kernels/coalesced.hpp"
#include "../expl_comp_strat/tpch_kit.hpp"
#include "../expl_comp_strat/common.hpp"

using std::make_pair;
using std::make_unique;
using std::unique_ptr;
using std::cout;
using std::endl;
using std::flush;

void syscall(std::string command) {
    auto x = system(command.c_str());
    (void) x;
}

#define GIGA (1024 * 1024 * 1024)
#define MEGA (1024 * 1024)
#define KILO (1024)

using timer = std::chrono::high_resolution_clock;

inline bool file_exists(const std::string& name) {
    struct stat buffer;
    return (stat (name.c_str(), &buffer) == 0);
}

inline std::string join_path(std::string a, std::string b) {
    return a + "/" + b;
}

std::ifstream::pos_type filesize(std::string filename) {
    std::ifstream in(filename.c_str(), std::ifstream::ate | std::ifstream::binary);
    return in.tellg();
}

// Note: This will force casts to int. It's not a problem
// the way our code is written, but otherwise it needs to be generalized
constexpr inline int div_rounding_up(const int& dividend, const int& divisor)
{
    // This is not the fastest implementation, but it's safe, in that there's never overflow
#if __cplusplus >= 201402L
    std::div_t div_result = std::div(dividend, divisor);
    return div_result.quot + !(!div_result.rem);
#else
    // Hopefully the compiler will optimize the two calls away.
    return std::div(dividend, divisor).quot + !(!std::div(dividend, divisor).rem);
#endif
}

template <typename UniquePtr>
void load_column_from_binary_file(
    UniquePtr&          buffer,
    cardinality_t       cardinality,
    const std::string&  directory,
    const std::string&  file_name)
{
    // TODO: C++'ify the file access (will also guarantee exception safety)
    using raw_ptr_type = typename std::decay<decltype(buffer.get())>::type;
    using element_type = typename std::remove_pointer<raw_ptr_type>::type;
    auto file_path = join_path(directory, file_name);
    buffer = std::make_unique<element_type[]>(cardinality);
    cout << "Loading a column from " << file_path << " ... " << flush;
    FILE* pFile = fopen(file_path.c_str(), "rb");
    if (pFile == nullptr) { throw std::runtime_error("Failed opening file " + file_path); }
    auto num_elements_read = fread(buffer.get(), sizeof(element_type), cardinality, pFile);
    if (num_elements_read != cardinality) {
        throw std::runtime_error("Failed reading sufficient data from " +
            file_path + " : expected " + std::to_string(cardinality) + " elements but read only " + std::to_string(num_elements_read) + "."); }
    fclose(pFile);
    cout << "done." << endl;
}

template <typename T>
void write_column_to_binary_file(const T* buffer, cardinality_t cardinality, const std::string& directory, const std::string& file_name) {
    auto file_path = join_path(directory, file_name);
    cout << "Writing a column to " << file_path << " ... " << flush;
    FILE* pFile = fopen(file_path.c_str(), "wb+");
    if (pFile == nullptr) { throw std::runtime_error("Failed opening file " + file_path); }
    auto num_elements_written = fwrite(buffer, sizeof(T), cardinality, pFile);
    fclose(pFile);
    if (num_elements_written != cardinality) {
        remove(file_path.c_str());
        throw std::runtime_error("Failed writing all elements to the file - only " +
            std::to_string(num_elements_written) + " written: " + strerror(errno));
    }
    cout << "done." << endl;
}

void print_help() {
    fprintf(stderr, "Unrecognized command line option.\n");
    fprintf(stderr, "Usage: tpch_01 [args]\n");
    fprintf(stderr, "   --scale_factor=[scale_factor:1] (number, e.g. 0.01 - 100)\n");
    fprintf(stderr, "   --streams=[streams:8] (number, e.g. 1 - 64)\n");
    fprintf(stderr, "   --tuples-per-stream=[tuples:32768] (number, e.g. 16384 - 131072)\n");
    fprintf(stderr, "   --values-per-thread=[values:64] (number, e.g. 16 - 128)\n");
    fprintf(stderr, "   --threads-per-block=[threads:32] (number, e.g. 32 - 1024)\n");
    fprintf(stderr, "   --use-global-ht\n");
    fprintf(stderr, "   --use-small-datatypes\n");
    fprintf(stderr, "   --use-coalescing\n");
}

template <typename F, typename... Args>
void for_each_argument(F f, Args&&... args) {
    [](...){}((f(std::forward<Args>(args)), 0)...);
}


int main(int argc, char** argv) {
    cout << "TPC-H Query 1" << '\n';
    /* load data */

    cardinality_t cardinality;

    double scale_factor = 1;
    int num_gpu_streams = defaults::num_gpu_streams;
    int num_threads_per_block = defaults::num_threads_per_block;
    int num_records_per_scheduled_kernel = defaults::num_records_per_scheduled_kernel;
        // Make sure it's a multiple of num_threads_per_block, or bad things may happen

    // This is the number of times we run the actual query execution - the part that we time;
    // it will not include initialization/allocations that are not necessary when the DBMS
    // is brought up. Note the allocation vs sub-allocation issue (see further comments below)
    int num_query_execution_runs = 5;

    std::string sf_argument = "--scale_factor=";
    std::string streams_argument = "--streams=";
    std::string threads_per_block_argument = "--threads-per-block=";
    std::string nruns_argument = "--runs=";
    for(int i = 1; i < argc; i++) {
        auto arg = std::string(argv[i]);
        if (arg == "--device") {
            get_device_properties();
            exit(1);
        } else if (arg.substr(0, sf_argument.size()) == sf_argument) {
            scale_factor = std::stod(arg.substr(sf_argument.size()));
        } else if (arg.substr(0, streams_argument.size()) == streams_argument) {
            num_gpu_streams = std::stoi(arg.substr(streams_argument.size()));
        } else if (arg.substr(0, threads_per_block_argument.size()) == threads_per_block_argument) {
            num_threads_per_block = std::stoi(arg.substr(threads_per_block_argument.size()));
        } else if (arg.substr(0, nruns_argument.size()) == nruns_argument) {
            num_query_execution_runs = std::stoi(arg.substr(nruns_argument.size()));
        } else {
            print_help();
            exit(1);
        }
    }
    lineitem li((size_t)(7000000 * scale_factor));
        // TODO: Make this magic number go away somehow; it's a maximum of 1,500,000 * 7

    std::unique_ptr< ship_date_t[]      > _shipdate;
    std::unique_ptr< return_flag_t[]    > _returnflag;
    std::unique_ptr< line_status_t[]    > _linestatus;
    std::unique_ptr< discount_t[]       > _discount;
    std::unique_ptr< tax_t[]            > _tax;
    std::unique_ptr< extended_price_t[] > _extendedprice;
    std::unique_ptr< quantity_t[]       > _quantity;

    // TODO: Use std::filesystem for the filesystem stuff
    syscall("mkdir -p tpch");
    std::string tpch_directory = join_path("tpch", std::to_string(scale_factor));
    syscall(std::string("mkdir -p ") + tpch_directory);
    if (file_exists(join_path(tpch_directory, "shipdate.bin"))) {
        // binary files (seem to) exist, load them
        cardinality = filesize(join_path(tpch_directory, "shipdate.bin")) / sizeof(ship_date_t);
        if (cardinality == 0) {
            throw std::runtime_error("The lineitem table column cardinality should not be 0");
        }
        load_column_from_binary_file(_shipdate,      cardinality, tpch_directory, "shipdate.bin");
        load_column_from_binary_file(_returnflag,    cardinality, tpch_directory, "returnflag.bin");
        load_column_from_binary_file(_linestatus,    cardinality, tpch_directory, "linestatus.bin");
        load_column_from_binary_file(_discount,      cardinality, tpch_directory, "discount.bin");
        load_column_from_binary_file(_tax,           cardinality, tpch_directory, "tax.bin");
        load_column_from_binary_file(_extendedprice, cardinality, tpch_directory, "extendedprice.bin");
        load_column_from_binary_file(_quantity,      cardinality, tpch_directory, "quantity.bin");
    } else {
        std::string input_file = join_path(tpch_directory, "lineitem.tbl");
        if (not file_exists(input_file.c_str())) {
            throw std::runtime_error("Cannot locate table text file " + input_file);
            // Not generating it ourselves - that's: 1. Not healthy and 2. Not portable;
            // setup scripts are intended to do that
        }
        cout << "Parsing the lineitem table in file " << input_file << endl;
        li.FromFile(input_file.c_str());
        cardinality = li.l_extendedprice.cardinality;
        if (cardinality == 0) {
            throw std::runtime_error("The lineitem table column cardinality should not be 0");
        }
        cout << "CSV read & parsed; table length: " << cardinality << " records." << endl;
        auto write_to = [&](auto& uptr, const char* filename) {
            using T = typename std::remove_pointer<typename std::decay<decltype(uptr.get())>::type>::type;
            load_column_from_binary_file(uptr.get(), cardinality, tpch_directory, "shipdate.bin");
        };
        write_column_to_binary_file(li.l_shipdate.get(),      cardinality, tpch_directory, "shipdate.bin");
        write_column_to_binary_file(li.l_returnflag.get(),    cardinality, tpch_directory, "returnflag.bin");
        write_column_to_binary_file(li.l_linestatus.get(),    cardinality, tpch_directory, "linestatus.bin");
        write_column_to_binary_file(li.l_discount.get(),      cardinality, tpch_directory, "discount.bin");
        write_column_to_binary_file(li.l_tax.get(),           cardinality, tpch_directory, "tax.bin");
        write_column_to_binary_file(li.l_extendedprice.get(), cardinality, tpch_directory, "extendedprice.bin");
        write_column_to_binary_file(li.l_quantity.get(),      cardinality, tpch_directory, "quantity.bin");
    }

    auto compressed_ship_date      = cuda::memory::host::make_unique< compressed::ship_date_t[]      >(cardinality);
    auto compressed_discount       = cuda::memory::host::make_unique< compressed::discount_t[]       >(cardinality);
    auto compressed_extended_price = cuda::memory::host::make_unique< compressed::extended_price_t[] >(cardinality);
    auto compressed_tax            = cuda::memory::host::make_unique< compressed::tax_t[]            >(cardinality);
    auto compressed_quantity       = cuda::memory::host::make_unique< compressed::quantity_t[]       >(cardinality);
    auto compressed_return_flag    = cuda::memory::host::make_unique< bit_container_t[] >(div_rounding_up(cardinality, return_flag_values_per_container));
    auto compressed_line_status    = cuda::memory::host::make_unique< bit_container_t[] >(div_rounding_up(cardinality, line_status_values_per_container));

    cout << "Preprocessing/compressing column data... " << flush;

    // Eyal says: Drop these copies, we really don't need them AFAICT
    auto shipdate      = _shipdate.get();
    auto returnflag    = _returnflag.get();
    auto linestatus    = _linestatus.get();
    auto discount      = _discount.get();
    auto tax           = _tax.get();
    auto extendedprice = _extendedprice.get();
    auto quantity      = _quantity.get();

    // Man, we really need to have a sub-byte-length-value container class
    std::memset(compressed_return_flag.get(), 0, div_rounding_up(cardinality, return_flag_values_per_container));
    std::memset(compressed_line_status.get(), 0, div_rounding_up(cardinality, line_status_values_per_container));
    for(cardinality_t i = 0; i < cardinality; i++) {
        compressed_ship_date[i]      = shipdate[i] - ship_date_frame_of_reference;
        compressed_discount[i]       = discount[i]; // we're keeping the factor 100 scaling
        compressed_extended_price[i] = extendedprice[i];
        compressed_quantity[i]       = quantity[i] / 100;
        compressed_tax[i]            = tax[i]; // we're keeping the factor 100 scaling
        set_bit_resolution_element<log_return_flag_bits, cardinality_t>(
            compressed_return_flag.get(), i, encode_return_flag(returnflag[i]));
        set_bit_resolution_element<log_line_status_bits, cardinality_t>(
            compressed_line_status.get(), i, encode_line_status(linestatus[i]));
        assert( (ship_date_t)      compressed_ship_date[i]      == shipdate[i] - ship_date_frame_of_reference);
        assert( (discount_t)       compressed_discount[i]       == discount[i]);
        assert( (extended_price_t) compressed_extended_price[i] == extendedprice[i]);
        assert( (quantity_t)       compressed_quantity[i]       == quantity[i] / 100);
            // not keeping the scaling here since we know the data is all integral; you could call this a form
            // of compression
        assert( (tax_t)            compressed_tax[i]            == tax[i]);
    }

    for(size_t i = 0; i < cardinality; i++) {
        assert(decode_line_status(get_bit_resolution_element<log_line_status_bits, cardinality_t>(compressed_line_status.get(), i)) == linestatus[i]);
        assert(decode_return_flag(get_bit_resolution_element<log_return_flag_bits, cardinality_t>(compressed_return_flag.get(), i)) == returnflag[i]);
    }
    cout << "done." << endl;

    // Note:
    // We are not timing the host-side allocations here. In a real DBMS, these will likely only be
    // a few sub-allocations, which would take very little time (dozens of clock cycles overall) -
    // no system calls.

    struct {
        std::unique_ptr<sum_quantity_t[]        > sum_quantity;
        std::unique_ptr<sum_base_price_t[]      > sum_base_price;
        std::unique_ptr<sum_discounted_price_t[]> sum_discounted_price;
        std::unique_ptr<sum_charge_t[]          > sum_charge;
        std::unique_ptr<sum_discount_t[]        > sum_discount;
        std::unique_ptr<cardinality_t[]         > record_count;
        // Why aren't we computing these?
        // struct {
        //     std::unique_ptr<avg_quantity_t[]        > avg_quantity;
        //     std::unique_ptr<avg_extended_price_t[]  > avg_extended_price;
        //     std::unique_ptr<avg_discount_t[]        > avg_discount;
        // } derived;
    } aggregates_on_host = {
        std::make_unique< sum_quantity_t[]         >(num_potential_groups),
        std::make_unique< sum_base_price_t[]       >(num_potential_groups),
        std::make_unique< sum_discounted_price_t[] >(num_potential_groups),
        std::make_unique< sum_charge_t []          >(num_potential_groups),
        std::make_unique< sum_discount_t[]         >(num_potential_groups),
        std::make_unique< cardinality_t[]          >(num_potential_groups)
        // ,
        // {
        //      std::make_unique< avg_quantity_t[]         >(num_potential_groups),
        //      std::make_unique< avg_extended_price_t[]   >(num_potential_groups),
        //      std::make_unique< avg_discount_t[]         >(num_potential_groups),
        // }
    };

    cuda::profiling::start();


    /* Allocate memory on device */

    // Note:
    // We are not timing the allocations here. In a real DBMS, actual CUDA allocations would
    // happen with the DBMS is brought up, and when a query is processed, it will only be
    // a few sub-allocations, which would take very little time (dozens of clock cycles overall) -
    // no CUDA API nor system calls. We _will_, however, time the initialization of the buffers.

    auto cuda_device = cuda::device::current::get();

    struct {
        cuda::memory::device::unique_ptr< sum_quantity_t[]         > sum_quantity;
        cuda::memory::device::unique_ptr< sum_base_price_t[]       > sum_base_price;
        cuda::memory::device::unique_ptr< sum_discounted_price_t[] > sum_discounted_price;
        cuda::memory::device::unique_ptr< sum_charge_t[]           > sum_charge;
        cuda::memory::device::unique_ptr< sum_discount_t[]         > sum_discount;
        cuda::memory::device::unique_ptr< cardinality_t[]          > record_count;
    } aggregates_on_device = {
        cuda::memory::device::make_unique< sum_quantity_t[]         >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_base_price_t[]       >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_discounted_price_t[] >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_charge_t []          >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< sum_discount_t[]         >(cuda_device, num_potential_groups),
        cuda::memory::device::make_unique< cardinality_t[]          >(cuda_device, num_potential_groups)
    };

    struct stream_input_buffer_set {
        cuda::memory::device::unique_ptr< compressed::ship_date_t[]      > shipdate;
        cuda::memory::device::unique_ptr< compressed::discount_t[]       > discount;
        cuda::memory::device::unique_ptr< compressed::extended_price_t[] > extendedprice;
        cuda::memory::device::unique_ptr< compressed::tax_t[]            > tax;
        cuda::memory::device::unique_ptr< compressed::quantity_t[]       > quantity;
        cuda::memory::device::unique_ptr< bit_container_t[]              > returnflag;
        cuda::memory::device::unique_ptr< bit_container_t[]              > linestatus;
    };

    std::vector<stream_input_buffer_set> stream_input_buffer_sets;
    std::vector<cuda::stream_t<>> streams;
    stream_input_buffer_sets.reserve(num_gpu_streams);
    streams.reserve(num_gpu_streams);
        // We'll be scheduling (most of) our work in a round-robin fashion on all of
        // the streams, to prevent the GPU from idling.


    for (int i = 0; i < num_gpu_streams; ++i) {
        auto input_buffers = stream_input_buffer_set{
            cuda::memory::device::make_unique< compressed::ship_date_t[]      >(cuda_device, num_records_per_scheduled_kernel),
            cuda::memory::device::make_unique< compressed::discount_t[]       >(cuda_device, num_records_per_scheduled_kernel),
            cuda::memory::device::make_unique< compressed::extended_price_t[] >(cuda_device, num_records_per_scheduled_kernel),
            cuda::memory::device::make_unique< compressed::tax_t[]            >(cuda_device, num_records_per_scheduled_kernel),
            cuda::memory::device::make_unique< compressed::quantity_t[]       >(cuda_device, num_records_per_scheduled_kernel),
            cuda::memory::device::make_unique< bit_container_t[]              >(cuda_device, div_rounding_up(num_records_per_scheduled_kernel, return_flag_values_per_container)),
            cuda::memory::device::make_unique< bit_container_t[]              >(cuda_device, div_rounding_up(num_records_per_scheduled_kernel, line_status_values_per_container))
        };
        stream_input_buffer_sets.emplace_back(std::move(input_buffers));
        auto stream = cuda_device.create_stream(cuda::stream::async);
        streams.emplace_back(std::move(stream));
    }

    // You can't measure this from inside the process - without events, which
//    double copy_time = 0;
//    double computation_time = 0;

    // This only works for the overall time, not for anything else, so it's not a good idea:
     std::ofstream results_file;
     results_file.open("results.csv", std::ios::out);

    for(int run_index = 0; run_index < num_query_execution_runs; run_index++) {
        cout << "Executing query, run " << run_index + 1 << " of " << num_query_execution_runs << "... " << flush;
        auto start = timer::now();

        // Initialize the aggregates; perhaps we should do this in a single kernel? ... probably not worth it
        streams[0].enqueue.memset(aggregates_on_device.sum_quantity.get(),         0, num_potential_groups * sizeof(sum_quantity_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_base_price.get(),       0, num_potential_groups * sizeof(sum_base_price_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_discounted_price.get(), 0, num_potential_groups * sizeof(sum_discounted_price_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_charge.get(),           0, num_potential_groups * sizeof(sum_charge_t));
        streams[0].enqueue.memset(aggregates_on_device.sum_discount.get(),         0, num_potential_groups * sizeof(sum_discount_t));
        streams[0].enqueue.memset(aggregates_on_device.record_count.get(),         0, num_potential_groups * sizeof(cardinality_t));

        cuda::event_t aggregates_initialized_event = cuda_device.create_event(
                cuda::event::sync_by_blocking, cuda::event::dont_record_timings, cuda::event::not_interprocess);
        streams[0].enqueue.event(aggregates_initialized_event);
        for (int i = 1; i < num_gpu_streams; ++i) {
            streams[i].enqueue.wait(aggregates_initialized_event);
            // The other streams also require the aggregates to be initialized before doing any work
        }
        auto stream_index = 0;
        for (size_t offset_in_table = 0;
             offset_in_table < cardinality;
             offset_in_table += num_records_per_scheduled_kernel,
             stream_index = (stream_index+1) % num_gpu_streams) {

            auto num_records_for_this_launch = std::min<cardinality_t>(num_records_per_scheduled_kernel, cardinality - offset_in_table);
            auto num_return_flag_bit_containers_for_this_launch = div_rounding_up(num_records_for_this_launch, return_flag_values_per_container);
            auto num_line_status_bit_containers_for_this_launch = div_rounding_up(num_records_for_this_launch, line_status_values_per_container);

            // auto start_copy = timer::now();  // This can't work, since copying is asynchronous.
            auto& stream = streams[stream_index];
            auto& stream_input_buffers = stream_input_buffer_sets[stream_index];
            stream.enqueue.copy(stream_input_buffers.shipdate.get()     , compressed_ship_date.get()      + offset_in_table, num_records_for_this_launch * sizeof(compressed::ship_date_t));
            stream.enqueue.copy(stream_input_buffers.discount.get()     , compressed_discount.get()       + offset_in_table, num_records_for_this_launch * sizeof(compressed::discount_t));
            stream.enqueue.copy(stream_input_buffers.extendedprice.get(), compressed_extended_price.get() + offset_in_table, num_records_for_this_launch * sizeof(compressed::extended_price_t));
            stream.enqueue.copy(stream_input_buffers.tax.get()          , compressed_tax.get()            + offset_in_table, num_records_for_this_launch * sizeof(compressed::tax_t));
            stream.enqueue.copy(stream_input_buffers.quantity.get()     , compressed_quantity.get()       + offset_in_table, num_records_for_this_launch * sizeof(compressed::quantity_t));
            stream.enqueue.copy(stream_input_buffers.returnflag.get()   , compressed_return_flag.get()    + offset_in_table / return_flag_values_per_container, num_return_flag_bit_containers_for_this_launch * sizeof(bit_container_t) );
            stream.enqueue.copy(stream_input_buffers.linestatus.get()   , compressed_line_status.get()    + offset_in_table / line_status_values_per_container, num_line_status_bit_containers_for_this_launch * sizeof(bit_container_t));

            auto num_blocks = div_rounding_up(num_records_for_this_launch, num_threads_per_block);
            auto launch_config = cuda::make_launch_config(num_blocks, num_threads_per_block);
            (void) launch_config;

            stream.enqueue.kernel_launch(
                cuda::thread_local_tpchQ01_small_datatypes_coalesced,
                launch_config,
                aggregates_on_device.sum_quantity.get(),
                aggregates_on_device.sum_base_price.get(),
                aggregates_on_device.sum_discounted_price.get(),
                aggregates_on_device.sum_charge.get(),
                aggregates_on_device.sum_discount.get(),
                aggregates_on_device.record_count.get(),
                stream_input_buffers.shipdate.get(),
                stream_input_buffers.discount.get(),
                stream_input_buffers.extendedprice.get(),
                stream_input_buffers.tax.get(),
                stream_input_buffers.quantity.get(),
                stream_input_buffers.returnflag.get(),
                stream_input_buffers.linestatus.get(),
                num_records_for_this_launch);

        }
        std::vector<cuda::event_t> completion_events;
        for(int i = 1; i < num_gpu_streams; i++) {
            auto event = streams[i].enqueue.event();
            completion_events.emplace_back(std::move(event));
        }

        // It's probably a better idea to go round-robin on the streams here
        streams[0].enqueue.copy(aggregates_on_host.sum_quantity.get(),         aggregates_on_device.sum_quantity.get(),         num_potential_groups * sizeof(sum_quantity_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_base_price.get(),       aggregates_on_device.sum_base_price.get(),       num_potential_groups * sizeof(sum_base_price_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_discounted_price.get(), aggregates_on_device.sum_discounted_price.get(), num_potential_groups * sizeof(sum_discounted_price_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_charge.get(),           aggregates_on_device.sum_charge.get(),           num_potential_groups * sizeof(sum_charge_t));
        streams[0].enqueue.copy(aggregates_on_host.sum_discount.get(),         aggregates_on_device.sum_discount.get(),         num_potential_groups * sizeof(sum_discount_t));
        streams[0].enqueue.copy(aggregates_on_host.record_count.get(),         aggregates_on_device.record_count.get(),         num_potential_groups * sizeof(cardinality_t));

        streams[0].synchronize();
        auto end = timer::now();
        std::chrono::duration<double> duration(end - start);
        cout << "done." << endl;
        results_file << duration.count() << '\n';
    }
    cuda::profiling::stop();

    if (num_query_execution_runs == 1) {
        cout << "\n"
                "+--------------------------------------------------- Results ---------------------------------------------------+\n";
        cout << "|  LS | RF | sum_quantity        | sum_base_price      | sum_disc_price      | sum_charge          | count      |\n";
        cout << "+---------------------------------------------------------------------------------------------------------------+\n";
        auto print_dec = [] (auto s, auto x) { printf("%s%16ld.%02ld", s, Decimal64::GetInt(x), Decimal64::GetFrac(x)); };

        for (size_t group=0; group<num_potential_groups; group++) {
            if (aggregates_on_host.record_count[group] > 0) {
                char rf = '-', ls = '-';
                auto return_flag_group_id = group >> 1;
                auto line_status_group_id = group & 0x1;
                switch(return_flag_group_id) {
                case 0:  rf = 'A'; break;
                case 1:  rf = 'F'; break;
                case 2:  rf = 'N'; break;
                default: rf = '-';
                }
                ls = (line_status_group_id == 0 ? 'F' : 'O');
                if (rf == 'A' and ls == 'F') {
                    if (cardinality == 6001215) {
                        assert(aggregates_on_host.sum_quantity[group] == 3773410700);
                        assert(aggregates_on_host.record_count[group] == 1478493);
                    }
                } else if (rf == 'N' and ls == 'F') {
                    if (cardinality == 6001215) {
                        assert(aggregates_on_host.sum_quantity[group] == 99141700);
                        assert(aggregates_on_host.record_count[group] == 38854);
                    }
                } else if (rf == 'N' and ls == 'O') {
                    rf = 'N';
                    ls = 'O';
                    if (cardinality == 6001215) {
                        assert(aggregates_on_host.sum_quantity[group] == 7447604000);
                        assert(aggregates_on_host.record_count[group] == 2920374);
                    }
                } else if (rf == 'R' and ls == 'F') {
                    if (cardinality == 6001215) {
                        assert(aggregates_on_host.sum_quantity[group]== 3771975300);
                        assert(aggregates_on_host.record_count[group]== 1478870);
                    }
                }

                printf("| # %c | %c ", rf, ls);
                print_dec(" | ",  aggregates_on_host.sum_quantity.get()[group]);
                print_dec(" | ",  aggregates_on_host.sum_base_price.get()[group]);
                print_dec(" | ",  aggregates_on_host.sum_discounted_price.get()[group]);
                print_dec(" | ",  aggregates_on_host.sum_charge.get()[group]);
                printf(" | %10u |\n", aggregates_on_host.record_count.get()[group]);
            }
        }

        cout << "+---------------------------------------------------------------------------------------------------------------+\n";
    }
    results_file.close();
}
